#include <emmintrin.h>
#include <sys/time.h> 
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <iostream>
#include <iomanip>
#include <fstream>

#include "GPUTSVAddPlugin.h"

void GPUTSVAddPlugin::input(std::string file) {
 inputfile = file;
 std::ifstream ifile(inputfile.c_str(), std::ios::in);
 while (!ifile.eof()) {
   std::string key, value;
   ifile >> key;
   ifile >> value;
   parameters[key] = value;
 }
 M = atoi(parameters["M"].c_str());
 N = atoi(parameters["N"].c_str());
 A = (double*) malloc(N*N*sizeof(double));
 B = (double*) malloc(N*N*sizeof(double));
 C = (double*) malloc(N*N*sizeof(double));
 std::ifstream myinput((std::string(PluginManager::prefix())+parameters["matrix1"]).c_str(), std::ios::in);
 int i;
 for (i = 0; i < M*N; ++i) {
	int k;
	myinput >> k;
        A[i] = k;
 }
 std::ifstream myinput2((std::string(PluginManager::prefix())+parameters["matrix2"]).c_str(), std::ios::in);
 for (i = 0; i < M*N; ++i) {
	int k;
	myinput2 >> k;
        B[i] = k;
 }
}




void GPUTSVAddPlugin::run() {
	double *pA;
	double *pB;
	double *pC;
hipMalloc((void**)&pA, (M*N)*sizeof(double));
hipMalloc((void**)&pB, (M*N)*sizeof(double));
hipMalloc((void**)&pC, (M*N)*sizeof(double));
hipMemcpy(pA, A, (M*N)*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(pB, B, (M*N)*sizeof(double), hipMemcpyHostToDevice);
printf("***Add on %d x %d Matrix on GPU***\n",N,N);
MatAdd<<<M,N>>>(pA, pB, pC, M, N);
hipMemcpy(C, pC, (M*N)*sizeof(double), hipMemcpyDeviceToHost);

hipFree(pA);
hipFree(pB);
hipFree(pC);

}

void GPUTSVAddPlugin::output(std::string file) {
	std::ofstream outfile(file.c_str(), std::ios::out);
        int i, j;
        for (i = 0; i < M; ++i){
            for (j = 0; j < N; ++j){
		outfile << C[i*N+j];//std::setprecision(0) << a[i*N+j];
		if (j != N-1)
			outfile << "\t";
		else
			outfile << "\n";
            }
	}
	free(A);
	free(B);
	free(C);
}



PluginProxy<GPUTSVAddPlugin> GPUTSVAddPluginProxy = PluginProxy<GPUTSVAddPlugin>("GPUTSVAdd", PluginManager::getInstance());


